#include "hip/hip_runtime.h"
#include "tensorkernels.cuh"


__global__ void fill_ones(float* M, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		M[idx] = 1.0f;
	}
}

__global__ void fill_zeros(float* M, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		M[idx] = 0.0f;
	}
}



void GPU_fillOnes(float* d_data, int size) {
	fill_ones << <128, 128 >> > (d_data, size);
}


void GPU_fillZeros(float* d_data, int size) {
	fill_zeros << <128, 128 >> > (d_data, size);
}